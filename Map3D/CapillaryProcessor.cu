#include "hip/hip_runtime.h"
#include "Utils.h"
#include "UtilsCUDA.h"
#include "CapillaryProcessor.h"

/*
	Kernel Device (GPU) variables and functions
	===========================================
*/

__global__ void applyHPF(byte* d_srcMatrix, byte* d_dstMatrix, int rows, int cols,
	size_t deepSmoothingKernelSize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x >= cols) || (y >= rows))
	{
		return;
	}

	const size_t halfKernelSize = deepSmoothingKernelSize / 2;

	// Skip margins with zeroing of result
	if ((x < halfKernelSize) ||
		(x >= cols - halfKernelSize) ||
		(y < halfKernelSize) ||
		(y >= rows - halfKernelSize))
	{
		d_dstMatrix[y * cols + x] = 0;
		return;
	}

	// Calculate excess over blurred
	unsigned int sum = 0;
	for (size_t kernelRow = y - halfKernelSize; kernelRow <= y + halfKernelSize; kernelRow++)
	{
		for (size_t kernelCol = x - halfKernelSize; kernelCol <= x + halfKernelSize; kernelCol++)
		{
			sum += d_srcMatrix[kernelRow * cols + kernelCol];
		}
	}
	float blurred = (float)sum / deepSmoothingKernelSize / deepSmoothingKernelSize;
	float excess = 2.0F * (d_srcMatrix[y * cols + x] / blurred - 0.75F);

	if (excess < 0.0F)
	{
		excess = 0.0F;
	}
	if (excess > 1.0F)
	{
		excess = 1.0F;
	}

	byte normalizedExcess = (byte)(WHITE * excess + 0.5F);
	d_dstMatrix[y * cols + x] = normalizedExcess;
}

/*
	Public Host (CPU) functions to call kernel Device (GPU) functions
	=================================================================
*/

CapillaryProcessor::CapillaryProcessor()
{
	m_fineSmoothingKernelSize = 0;
	m_deepSmoothingKernelSize = 0;
	m_numDescribedCappilaries = 0;
	m_minPixelsInCappilary = 0;
	m_surroundingPixels = 0;

	m_originalMatrix = ByteMatrix();
	m_processedMatrix = ByteMatrix();
	m_layerIndex = 0;
}

void CapillaryProcessor::init(Config& config)
{
	initConfig(config);
}

void CapillaryProcessor::describeCapillaries(Map& map, LayerInfo& layerInfo, const std::string& outputFolderName)
{
	m_layerIndex = layerInfo.layerIndex;

	// Reset scores given by corner detection - new scores will be given by pixels in frame
	layerInfo.maxScore = 0.0F;
	layerInfo.sumScore = 0.0F;

	// Create folder for layer with the corresponding name
	std::string layerFolderName = "Layer" + std::to_string(layerInfo.layerIndex + 1);
	createFoldersIfNeed(outputFolderName, layerFolderName);

	// Get pixel matrix of current layer from the map
	Layer layer = map.getLayers()[layerInfo.layerIndex];
	m_originalMatrix = layer.matrix;
#ifdef _DEBUG
	cv::imwrite(outputFolderName + "/" + layerFolderName + "/Original.bmp", m_originalMatrix.asCvMatU8());
#endif
	// Create and fill processed matrix of current layer
	m_processedMatrix = ByteMatrix(m_originalMatrix.rows(), m_originalMatrix.cols());

	// Members passed as parameters to support filtering in chain
	performExcessFiltering(m_originalMatrix, m_processedMatrix);
#ifdef _DEBUG
	cv::imwrite(outputFolderName + "/" + layerFolderName + "/Processed.bmp", m_processedMatrix.asCvMatU8());
#endif
	size_t numOfDescribedCapillaries = layerInfo.capillaryApexes.size();

	std::cout << "Layer " << m_layerIndex + 1 << " - describing of capillaries started: " <<
		numOfDescribedCapillaries << " capillaries" << std::endl;
	m_timer.start();

	// For each capillary in the layer: calculate and collect information about the capillary
	for (size_t capillaryIndex = 0; capillaryIndex < numOfDescribedCapillaries; capillaryIndex++)
	{
		// Get coordinates of detected point in the capillary
		ScoredCorner scoredCorner = layerInfo.capillaryApexes[capillaryIndex];

		// Set initial information of the capillary
		CapillaryInfo capillaryInfo;
		capillaryInfo.index = capillaryIndex;
		capillaryInfo.setPos(scoredCorner);

		// Convert position of the corner from mm to pixels
		size_t cornerRow = mm2pixels(scoredCorner.y);
		size_t cornerCol = mm2pixels(scoredCorner.x);

		// Mark pixels of the capillary and update information
		performTraversalBFS(cornerRow, cornerCol, map, capillaryInfo);

		// Skip too sparse capillary
		if (capillaryInfo.pixelsCapillary < m_minPixelsInCappilary)
		{
			continue;
		}

		std::string info = "Capillary " + std::to_string(capillaryIndex + 1) + ": " +
			std::to_string(capillaryInfo.pixelsCapillary) + " pixels";

		// Skip too low or narrow capillary
		size_t capillaryRows = capillaryInfo.limitDn - capillaryInfo.limitUp + 1;
		size_t capillaryCols = capillaryInfo.limitRt - capillaryInfo.limitLf + 1;
		if ((capillaryRows < FRAME_HEIGHT) || (capillaryCols < FRAME_WIDTH))
		{
			info += " - too small";
			std::cout << info << std::endl;
			continue;
		}

		// Instance to find inscribed rotated frame
		MaxRectangle maxRectangleFinder(m_processedMatrix,
			PixelPos(capillaryInfo.limitUp, capillaryInfo.limitLf),
			capillaryRows, capillaryCols, outputFolderName + "/" + layerFolderName, capillaryIndex);

		// Rotated frame - is empty if cannot find rectangle with score over the threshold
		std::vector<PixelPos> rotatedRectangle =
			maxRectangleFinder.findRectangle(outputFolderName + "/" + layerFolderName, capillaryIndex);

		// Skip capillary with score lower than the threshold for which no frame was found
		if (rotatedRectangle.empty())
		{
			info += " - score is lower than threshold";
			std::cout << info << std::endl;
			continue;
		}

		// Angle of frame rotation
		capillaryInfo.angle = maxRectangleFinder.getAngle();

		// Score indicated percentage of marked pixels in the frame
		capillaryInfo.score = maxRectangleFinder.getScore();

		info += " - score = " + toString(capillaryInfo.score, 1);
		std::cout << info << std::endl;

		// Update description of capillaries for further statistics and score calculation
		layerInfo.capillariesInfo.push_back(capillaryInfo);
#ifdef _DEBUG
		drawRotatedFrame(rotatedRectangle);
#endif
	}

	m_timer.end();
	std::cout << "Layer " << m_layerIndex + 1 <<
		" - describing of capillaries completed in " <<
		m_timer.getDurationMilliseconds() << " ms" << std::endl << std::endl;
#ifdef _DEBUG
	cv::imwrite(outputFolderName + "/" + layerFolderName + "/Marked.bmp", m_processedMatrix.asCvMatU8());
#endif
	if (layerInfo.capillariesInfo.empty())
	{
		std::cout << "Layer " << m_layerIndex + 1 <<
			" - no capillaries found to hold FOV frame" << std::endl << std::endl;
		return;
	}

	float startXmm = map.getStartXmm();
	float startYmm = map.getStartYmm();
	collectSurroundings(layerInfo.capillariesInfo);
	trimAndSetLayerScores(layerInfo, startXmm, startYmm,
		layerInfo.capillariesInfo, outputFolderName + "/" + layerFolderName);
#ifdef _DEBUG
	cv::imwrite(outputFolderName + "/" + layerFolderName + "/Framed.bmp", m_originalMatrix.asCvMatU8());
#endif
}

/*
	Private Host (CPU) functions to call kernel Device (GPU) functions
	==================================================================
*/

void CapillaryProcessor::initConfig(Config& config)
{
	// Get parameters from configuration
	m_fineSmoothingKernelSize	= (size_t)config.getIntValue(keyFineSmoothingKernelSize);
	m_deepSmoothingKernelSize	= (size_t)config.getIntValue(keyDeepSmoothingKernelSize);
	m_numDescribedCappilaries	= (size_t)config.getIntValue(keyNumDescribedCappilaries);
	m_minPixelsInCappilary		= (size_t)config.getIntValue(keyMinPixelsInCappilary);
	m_surroundingPixels			= (size_t)config.getIntValue(keySurroundingPixels);
}

void CapillaryProcessor::performGaussianBlur(ByteMatrix& src, ByteMatrix& dst)
{
	size_t rows = src.rows();
	size_t cols = src.cols();

	// For each row in the source matrix
	for (size_t row = 0; row < rows; row++)
	{
		// For each col in the source matrix
		for (size_t col = 0; col < cols; col++)
		{
			// Skip the margin of source - copy source pixels to destination
			if ((row == 0) || (row == rows - 1) || (col == 0) || (col == cols - 1))
			{
				dst.set(row, col, src.get(row, col));
				continue;
			}

			// Calculate Gaussian blur
			unsigned short sum = 4 * src.get(row, col) +
				2 * (src.get(row, col - 1) + src.get(row, col + 1) + src.get(row - 1, col) + src.get(row + 1, col)) +
				src.get(row - 1, col - 1) + src.get(row - 1, col + 1) + src.get(row + 1, col - 1) + src.get(row + 1, col + 1);
			byte blurredPixel = (byte)std::roundf((float)sum / 16);
			dst.set(row, col, blurredPixel);
		}
	}
}

void CapillaryProcessor::performUniformSmoothing(ByteMatrix& src, ByteMatrix& dst)
{
	size_t rows = src.rows();
	size_t cols = src.cols();

	const size_t halfKernelSize = m_fineSmoothingKernelSize / 2;

	// For each row in the source matrix
	for (size_t row = 0; row < rows; row++)
	{
		// For each col in the source matrix
		for (size_t col = 0; col < cols; col++)
		{
			// Skip the margin of source - copy source pixels to destination
			if ((row < halfKernelSize) || (row >= rows - halfKernelSize) ||
				(col < halfKernelSize) || (col >= cols - halfKernelSize))
			{
				dst.set(row, col, src.get(row, col));
				continue;
			}

			// Calculate and set smoothed gray level
			unsigned int sum = 0;
			for (size_t kernelRow = row - halfKernelSize; kernelRow <= row + halfKernelSize; kernelRow++)
			{
				for (size_t kernelCol = col - halfKernelSize; kernelCol <= col + halfKernelSize; kernelCol++)
				{
					sum += src.get(kernelRow, kernelCol);
				}
			}
			float smoothed = (float)sum / m_fineSmoothingKernelSize / m_fineSmoothingKernelSize;
			byte smoothedPixel = (byte)std::roundf(smoothed);
			dst.set(row, col, smoothedPixel);
		}
	}
}

void CapillaryProcessor::performExcessFiltering(ByteMatrix& src, ByteMatrix& dst)
{
	size_t rows = src.rows();
	size_t cols = src.cols();

	// Parameters to launch parallel threads
	dim3 blockSize(128, 1);
	dim3 numBlocks(divideCeil((int)cols, blockSize.x), divideCeil((int)rows, blockSize.y), 1);

	std::cout << "Layer " << m_layerIndex + 1 <<
		" - applying of excess HPF started" << std::endl;
	m_timer.start();

	// Allocate device memory buffers and fill source device buffer by processed matrix
	byte* d_srcBuffer = nullptr;
	byte* d_dstBuffer = nullptr;
	checkCuda(hipMalloc(&d_srcBuffer, rows * cols));
	checkCuda(hipMalloc(&d_dstBuffer, rows * cols));
	checkCuda(hipMemcpy(d_srcBuffer, src.getBuffer(), rows * cols, hipMemcpyHostToDevice));

	// Calculate excess on GPU
	applyHPF<<<numBlocks, blockSize>>>(d_srcBuffer, d_dstBuffer, (int)rows, (int)cols,
		m_deepSmoothingKernelSize);
	checkCuda(hipDeviceSynchronize());

	// Get calculated excess from device memory and free it
	checkCuda(hipMemcpy(dst.getBuffer(), d_dstBuffer, rows * cols, hipMemcpyDeviceToHost));
	checkCuda(hipFree(d_srcBuffer));
	checkCuda(hipFree(d_dstBuffer));

	m_timer.end();
	std::cout << "Layer " << m_layerIndex + 1 <<
		" - applying of excess HPF completed in " <<
		m_timer.getDurationMilliseconds() << " ms" << std::endl;
}

/*
	Perform traversal of connected pixels in the area from given root: row and col.
	Pixels in the area assumed have valid gray level which means predefined interval.
	The area traversal is performed using the BFS (Breadth-First Search) algorithn.
	This algorithm is based on non-recursive iteration in loop with usage of queue.
	Recursive traversal with the DFS (Depth-First Search) leads to stack overflow.
*/
void CapillaryProcessor::performTraversalBFS(size_t row, size_t col, Map& map, CapillaryInfo& capillaryInfo)
{
	// Define queue of pixel positions for traversal by BFS algorithn
	std::queue<PixelPos> pixels;

	// Root pixel is always with valid gray level
	PixelPos pixelRoot = PixelPos(row, col);
	pixels.push(pixelRoot);

	// Init upper and left limits for further minimization
	capillaryInfo.limitUp = m_processedMatrix.rows();
	capillaryInfo.limitLf = m_processedMatrix.cols();

	// The queue of pixels dynamically shrinks and grows on each cycle of the loop
	while (!pixels.empty())
	{
		// Take pixel from the front of queue and dequeue it as (already) processed
		PixelPos pixelPos = pixels.front();
		pixels.pop();

		// Skip already processed pixel - it occurs with the same directions in different order
		if (m_processedMatrix.get(pixelPos.pixelRow, pixelPos.pixelCol) == WHITE)
		{
			continue;
		}

		// Do not process pixels on seams but continue the traversal
		if (map.isOnSeam(pixelPos.pixelRow, true) || map.isOnSeam(pixelPos.pixelCol, false))
		{
			continue;
		}

		// Process the pixel and mark as already processed
		processPixel(pixelPos, capillaryInfo);

		// Enqueue the pixel from (row - 1) if row is in limits and gray level of the pixel is valid
		if ((pixelPos.pixelRow > m_deepSmoothingKernelSize) &&
			isValidGrayLevelProcessed(m_processedMatrix.get(pixelPos.pixelRow - 1, pixelPos.pixelCol)))
		{
			pixels.push(PixelPos(pixelPos.pixelRow - 1, pixelPos.pixelCol));
		}

		// Enqueue the pixel from (row + 1) if row is in limits and gray level of the pixel is valid
		if ((pixelPos.pixelRow < m_processedMatrix.rows() - m_deepSmoothingKernelSize) &&
			isValidGrayLevelProcessed(m_processedMatrix.get(pixelPos.pixelRow + 1, pixelPos.pixelCol)))
		{
			pixels.push(PixelPos(pixelPos.pixelRow + 1, pixelPos.pixelCol));
		}

		// Enqueue the pixel from (col - 1) if col is in limits and gray level of the pixel is valid
		if ((pixelPos.pixelCol > m_deepSmoothingKernelSize) &&
			isValidGrayLevelProcessed(m_processedMatrix.get(pixelPos.pixelRow, pixelPos.pixelCol - 1)))
		{
			pixels.push(PixelPos(pixelPos.pixelRow, pixelPos.pixelCol - 1));
		}

		// Enqueue the pixel from (col + 1) if col is in limits and gray level of the pixel is valid
		if ((pixelPos.pixelCol < m_processedMatrix.cols() - m_deepSmoothingKernelSize) &&
			isValidGrayLevelProcessed(m_processedMatrix.get(pixelPos.pixelRow, pixelPos.pixelCol + 1)))
		{
			pixels.push(PixelPos(pixelPos.pixelRow, pixelPos.pixelCol + 1));
		}
	}
}

void CapillaryProcessor::processPixel(const PixelPos& pixelPos, CapillaryInfo& capillaryInfo)
{
	// Update limits if processed pixel exceeds existing
	capillaryInfo.limitUp = std::min(capillaryInfo.limitUp, pixelPos.pixelRow);
	capillaryInfo.limitDn = std::max(capillaryInfo.limitDn, pixelPos.pixelRow);
	capillaryInfo.limitLf = std::min(capillaryInfo.limitLf, pixelPos.pixelCol);
	capillaryInfo.limitRt = std::max(capillaryInfo.limitRt, pixelPos.pixelCol);

	// Accumulate number of pixels and sum of gray levels in the capillary on original matrix
	capillaryInfo.pixelsCapillary++;
	capillaryInfo.energyCapillary += m_originalMatrix.get(pixelPos.pixelRow, pixelPos.pixelCol);

	// Mark the pixel as already processed
	m_processedMatrix.set(pixelPos.pixelRow, pixelPos.pixelCol, WHITE);
}

void CapillaryProcessor::collectSurroundings(std::vector<CapillaryInfo>& capillariesInfo)
{
	// Accumulate number of pixels and sum of gray levels in surrounding rectangles of each capillary
	for (CapillaryInfo& capillaryInfo : capillariesInfo)
	{
		if (capillaryInfo.pixelsCapillary < m_minPixelsInCappilary)
		{
			continue;
		}

		// Rectangle at up
		size_t rectUpUp = std::max((int)capillaryInfo.limitUp - (int)m_surroundingPixels, 0);
		size_t rectUpDn = capillaryInfo.limitUp;
		size_t rectUpLf = capillaryInfo.limitLf;
		size_t rectUpRt = capillaryInfo.limitRt;
		updateSurroundingData(capillaryInfo, rectUpUp, rectUpDn, rectUpLf, rectUpRt);

		// Rectangle at down
		size_t rectDnUp = capillaryInfo.limitDn;
		size_t rectDnDn = std::min((int)capillaryInfo.limitDn + (int)m_surroundingPixels, (int)m_processedMatrix.rows());
		size_t rectDnLf = capillaryInfo.limitLf;
		size_t rectDnRt = capillaryInfo.limitRt;
		updateSurroundingData(capillaryInfo, rectDnUp, rectDnDn, rectDnLf, rectDnRt);

		// Rectangle at left
		size_t rectLfUp = capillaryInfo.limitUp;
		size_t rectLfDn = capillaryInfo.limitDn;
		size_t rectLfLf = std::max((int)capillaryInfo.limitLf - (int)m_surroundingPixels, 0);
		size_t rectLfRt = capillaryInfo.limitLf;
		updateSurroundingData(capillaryInfo, rectLfUp, rectLfDn, rectLfLf, rectLfRt);

		// Rectangle at right
		size_t rectRtUp = capillaryInfo.limitUp;
		size_t rectRtDn = capillaryInfo.limitDn;
		size_t rectRtLf = capillaryInfo.limitRt;
		size_t rectRtRt = std::min((int)capillaryInfo.limitRt + (int)m_surroundingPixels, (int)m_processedMatrix.cols());
		updateSurroundingData(capillaryInfo, rectRtUp, rectRtDn, rectRtLf, rectRtRt);
	}
}

void CapillaryProcessor::updateSurroundingData(CapillaryInfo& capillaryInfo,
	size_t rectUp, size_t rectDn, size_t rectLf, size_t rectRt)
{
	capillaryInfo.pixelsSurroundings += (rectDn - rectUp) * (rectRt - rectLf);
	for (size_t row = rectUp; row < rectDn; row++)
	{
		for (size_t col = rectLf; col < rectRt; col++)
		{
			capillaryInfo.energySurroundings += m_processedMatrix.get(row, col);
		}
	}
}

void CapillaryProcessor::trimAndSetLayerScores(LayerInfo& layerInfo, float startXmm, float startYmm,
	std::vector<CapillaryInfo>& capillariesInfo, const std::string& layerFolderName)
{
	// Sort found capillaries by score in descending order
	std::sort(capillariesInfo.begin(), capillariesInfo.end(), [](CapillaryInfo capillaryL, CapillaryInfo capillaryR) {
		return capillaryL.score > capillaryR.score;
		});
	layerInfo.maxScore = capillariesInfo.begin()->score;

	// Keep only predefined number of capillaries with best scores
	if (capillariesInfo.size() > m_numDescribedCappilaries)
	{
		capillariesInfo.erase(capillariesInfo.begin() + m_numDescribedCappilaries, capillariesInfo.end());
	}

	std::string filenameData = layerFolderName + "/Data.csv";
	std::ofstream fileData(filenameData);
	fileData << "Num,x (col),y (row),z,Angle rad,Contrast,Score" << std::endl;

	// For each capillary among predefined number of capillaries with best scores
	for (const CapillaryInfo& capillaryInfo : capillariesInfo)
	{
		// Calculate contrast
		byte contrast = 0;
		if (capillaryInfo.pixelsCapillary >= m_minPixelsInCappilary)
		{
			byte avgGrayLevelCapillary =
				(byte)std::roundf((float)capillaryInfo.energyCapillary / capillaryInfo.pixelsCapillary);
			byte avgGrayLevelSurroundings =
				(byte)std::roundf((float)capillaryInfo.energySurroundings / capillaryInfo.pixelsSurroundings);
			contrast = avgGrayLevelSurroundings - avgGrayLevelCapillary;
		}

		fileData <<
			capillaryInfo.index + 1 << "," <<
			std::setprecision(4) << capillaryInfo.posApex.x + startXmm << " (" << mm2pixels(capillaryInfo.posApex.x) << ")," <<
			std::setprecision(4) << capillaryInfo.posApex.y + startYmm << " (" << mm2pixels(capillaryInfo.posApex.y) << ")," <<
			std::setprecision(4) << capillaryInfo.posApex.z << "," <<
			std::setprecision(2) << capillaryInfo.angle << "," <<
			(int)contrast << "," <<
			std::setprecision(2) << capillaryInfo.score << std::endl;

		layerInfo.sumScore += capillaryInfo.score;
	}

	fileData.close();
}

void CapillaryProcessor::drawRotatedFrame(const std::vector<PixelPos>& rotatedFrame)
{
	drawLine(rotatedFrame[0], rotatedFrame[1]);
	drawLine(rotatedFrame[1], rotatedFrame[2]);
	drawLine(rotatedFrame[2], rotatedFrame[3]);
	drawLine(rotatedFrame[3], rotatedFrame[0]);
}

void CapillaryProcessor::drawLine(PixelPos pixelA, PixelPos pixelB)
{
	PixelPos pixelBegin;
	PixelPos pixelEnd;

	// Check whether loop iteration is performed along X or Y
	if (std::abs((int)pixelA.pixelCol - (int)pixelB.pixelCol) >=
		std::abs((int)pixelA.pixelRow - (int)pixelB.pixelRow))
	{
		if (pixelA.pixelCol < pixelB.pixelCol)
		{
			pixelBegin = pixelA;
			pixelEnd = pixelB;
		}
		else
		{
			pixelBegin = pixelB;
			pixelEnd = pixelA;
		}
		float slope = ((float)pixelEnd.pixelRow - (float)pixelBegin.pixelRow) /
			((float)pixelEnd.pixelCol - (float)pixelBegin.pixelCol);
		for (size_t col = pixelBegin.pixelCol; col <= pixelEnd.pixelCol; col++)
		{
			size_t row = pixelBegin.pixelRow +
				(size_t)std::roundf(slope * (col - pixelBegin.pixelCol));
			m_originalMatrix.set(row, col, WHITE);
		}
	}
	else
	{
		if (pixelA.pixelRow < pixelB.pixelRow)
		{
			pixelBegin = pixelA;
			pixelEnd = pixelB;
		}
		else
		{
			pixelBegin = pixelB;
			pixelEnd = pixelA;
		}
		float slope = ((float)pixelEnd.pixelCol - (float)pixelBegin.pixelCol) /
			((float)pixelEnd.pixelRow - (float)pixelBegin.pixelRow);
		for (size_t row = pixelBegin.pixelRow; row <= pixelEnd.pixelRow; row++)
		{
			size_t col = pixelBegin.pixelCol +
				(size_t)std::roundf(slope * (row - pixelBegin.pixelRow));
			m_originalMatrix.set(row, col, WHITE);
		}
	}
}
