#include "hip/hip_runtime.h"
#include <regex>
#include <iterator>

#include "Utils.h"
#include "UtilsCUDA.h"
#include "Map.h"

/*
	Kernel Device (GPU) variables and functions
	===========================================
*/

/*
	Public Host (CPU) functions to call kernel Device (GPU) functions
	=================================================================
*/

Map::Map()
{
	m_startXmm = 0.0F;
	m_startYmm = 0.0F;
	m_stepXmm = 0.0F;
	m_stepYmm = 0.0F;
	m_rows = 0;
	m_cols = 0;
	m_markerSize = 21;
}

void Map::buildMap(const std::string& folderName)
{
#ifdef _DEBUG
	std::string config = "DEBUG";
#else
	std::string config = "RELEASE";
#endif
	std::cout << "Build 3D map in " << config << " configuration" << std::endl << std::endl;
	std::string absFolderName = getAbsFolderName(folderName);
	std::cout << "Input data folder:" << std::endl << absFolderName << std::endl << std::endl;

	// Vector of X-Y-Z coordinates
	std::vector<std::vector<std::string>> scanPositions = readScanPositions(folderName);

	// Images in the order of above scan positions
	std::cout << "Start loading of " << scanPositions[0].size() << " images" << std::endl;
	m_timer.start();
	std::vector<cv::Mat> images = readImages(folderName);
	m_timer.end();
	std::cout << "Images are loaded in " <<
		m_timer.getDurationMilliseconds() << " ms" << std::endl << std::endl;

	// Allocate byte matrices on each layer
	initLayers(images[0]);

	// Build stitched images on each layer
	std::cout << "Start stitching of " << scanPositions[0].size() << " images" << std::endl;
	m_timer.start();
	stitchImages(scanPositions, images);
	m_timer.end();
	std::cout << "Images are stitched in " <<
		m_timer.getDurationMilliseconds() << " ms" << std::endl << std::endl;
}

void Map::printValueAtTruncatedPos(float x, float y, float z)
{
	std::cout << "Get value on some position truncated to scan grid" << std::endl;

	size_t layersNum = m_layers.size();

	// Validate z
	if (z < m_layers[0].z)
	{
		std::cout << "Position z is less than lower bound" << std::endl;
	}
	if (z > m_layers[layersNum - 1].z)
	{
		std::cout << "Position z is more than upper bound" << std::endl;
	}

	// Find index of the layer for which given z is between iterated and next layer
	size_t layerIndex = 0;
	while (layerIndex < layersNum - 2)
	{
		if ((m_layers[layerIndex].z <= z) && (m_layers[layerIndex + 1].z > z))
		{
			break;
		}
		layerIndex++;
	}

	// Find col that matches truncated x
	float minX = m_indexedPositionsX.begin()->first;
	if (x < minX)
	{
		std::cout << "Position x is less than lower bound" << std::endl;
	}
	size_t col = (size_t)floor((x - minX) / m_stepXmm);
	if (col > m_cols - 1)
	{
		std::cout << "Position x is more than upper bound" << std::endl;
	}

	// Find row that matches truncated y
	float minY = m_indexedPositionsY.begin()->first;
	if (y < minY)
	{
		std::cout << "Position y is less than lower bound" << std::endl;
	}
	size_t row = (size_t)floor((y - minY) / m_stepYmm);
	if (row > m_rows - 1)
	{
		std::cout << "Position y is more than upper bound" << std::endl;
	}

	byte val = m_layers[layerIndex].matrix.get(row, col);
	std::cout << "x = " << x << std::endl << "y = " << y << std::endl << "z = " << z << std::endl <<
		"value = " << (int)val << std::endl << std::endl;
}

void Map::saveStiched(std::vector<LayerInfo>& layersWithCapillaries, const std::string& outputFolderName)
{
	createFoldersIfNeed(outputFolderName, "Stitched");
	size_t layersNum = m_layers.size();
	std::cout << "Start saving of stitched images on " << layersNum << " layers" << std::endl;
	m_timer.start();

	if (layersWithCapillaries.empty())
	{
		// Called before capillaries detection: save all layers without detectied capillaries
		for (size_t layerIndex = 0; layerIndex < layersNum; layerIndex++)
		{
			ByteMatrix layerMatrix = m_layers[layerIndex].matrix;
			std::string layerFilename = outputFolderName + "/Stitched/Layer" +
				std::to_string(layerIndex + 1) + ".bmp";
			bool result = cv::imwrite(layerFilename, layerMatrix.asCvMatU8());
			if (!result)
			{
				throw std::exception(("Cannot write file: " + layerFilename).c_str());
			}
		}
	}
	else
	{
		// Called after capillaries detection: save actual layers with detectied capillaries
		for (const LayerInfo& layerInfo : layersWithCapillaries)
		{
			ByteMatrix layerMatrix = m_layers[layerInfo.layerIndex].matrix;
			std::vector<ScoredCorner> scoredCorners = layerInfo.capillaryApexes;
			markCorners(layerMatrix, scoredCorners);
			std::string layerFilename = outputFolderName + "/Stitched/LayerDetected" +
				std::to_string(layerInfo.layerIndex + 1) + ".bmp";
			bool result = cv::imwrite(layerFilename, layerMatrix.asCvMatU8());
			if (!result)
			{
				throw std::exception(("Cannot write file: " + layerFilename).c_str());
			}
		}
	}

	m_timer.end();
	std::cout << "Stitched images are saved in " << m_timer.getDurationMilliseconds() << " ms" << std::endl << std::endl;
}

bool Map::isOnSeam(size_t posPixels, bool isRow)
{
	return isRow ?
		std::find(m_seamRows.begin(), m_seamRows.end(), posPixels) != m_seamRows.end() :
		std::find(m_seamCols.begin(), m_seamCols.end(), posPixels) != m_seamCols.end();
}

std::vector<Layer> Map::getLayers()
{
	return m_layers;
}

float Map::getStartXmm()
{
	return m_startXmm;
}

float Map::getStartYmm()
{
	return m_startYmm;
}

/*
	Private Host (CPU) functions to call kernel Device (GPU) functions
	==================================================================
*/

std::vector<std::vector<std::string>> Map::readScanPositions(const std::string& folderName)
{
	// Open file with scan positions
	std::string scanPosPathFilename = folderName + "/" + SCAN_POS_FILENAME;
	std::ifstream scanPosFile(scanPosPathFilename);

	// Parse scan positions into intermediate array of X-Y-Z coordinates
	std::string line;
	const std::regex comma(",");
	size_t scansNum = 0;
	std::vector<std::vector<std::string>> scanPositions;
	while (!scanPosFile.eof())
	{
		getline(scanPosFile, line);
		if (line.empty())
		{
			break;
		}
		std::sregex_token_iterator tokenIterator(line.begin(), line.end(), comma, -1);
		std::vector<std::string> coords{ tokenIterator, std::sregex_token_iterator() };
		scanPositions.push_back(coords);
		if (scansNum == 0)
		{
			scansNum = coords.size();
		}
		else
		{
			if (scansNum != coords.size())
			{
				throw std::exception("Mismatch number of coordinates");
			}
		}
	}

	// Get unique scan positions in all X-Y-Z coordinates with sequential indexes
	m_indexedPositionsX = getUniqueIndexedPositions(scanPositions[0]);
	m_indexedPositionsY = getUniqueIndexedPositions(scanPositions[1]);
	m_indexedPositionsZ = getUniqueIndexedPositions(scanPositions[2]);

	// Calculate the step in mm by X as difference of sequential X positions
	std::map<float, size_t>::iterator itrPositionsX = m_indexedPositionsX.begin();
	float initX = itrPositionsX->first;
	itrPositionsX++;
	float nextX = itrPositionsX->first;
	m_stepXmm = nextX - initX;

	// Calculate the step in mm by Y as difference of sequential Y positions
	std::map<float, size_t>::iterator itrPositionsY = m_indexedPositionsY.begin();
	float initY = itrPositionsY->first;
	itrPositionsY++;
	float nextY = itrPositionsY->first;
	m_stepYmm = nextY - initY;

	return scanPositions;
}

std::map<float, size_t> Map::getUniqueIndexedPositions(const std::vector<std::string>& coords)
{
	std::vector<float> uniquePositions;
	for (std::string coord : coords)
	{
		float val = (float)atof(coord.c_str());
		if (std::find(uniquePositions.begin(), uniquePositions.end(), val) == uniquePositions.end())
		{
			uniquePositions.push_back(val);
		}
	}
	std::sort(uniquePositions.begin(), uniquePositions.end());

	std::map<float, size_t> indexedPositions;
	size_t index = 0;
	for (float uniquePosition : uniquePositions)
	{
		indexedPositions[uniquePosition] = index++;
	}

	return indexedPositions;
}

std::vector<cv::Mat> Map::readImages(const std::string& folderName)
{
	size_t filesNum = getFilesNum(folderName);

	const size_t filenameSize = 32;
	char filename[filenameSize];
	std::vector<cv::Mat> images;
	for (size_t fileIndex = 0; fileIndex < filesNum; fileIndex++)
	{
		sprintf_s(filename, filenameSize, "Bright%4d.tif", (int)fileIndex);
		std::string pathFilename = folderName + "/" + filename;
		cv::Mat image = cv::imread(pathFilename, cv::IMREAD_GRAYSCALE);

		images.push_back(image);
		if ((fileIndex > 0) && (fileIndex % 20 == 0))
		{
			std::cout << "Loaded " << std::setw(3) << fileIndex << " images" << std::endl;
		}
	}

	return images;
}

void Map::initLayers(const cv::Mat& firstImage)
{
	m_cols = (size_t)((mm2pixels(m_stepXmm) + BIAS_X_PIXELS) * (m_indexedPositionsX.size() - 1) +
		FRAME_REL_W * firstImage.cols);
	m_rows = (size_t)(mm2pixels(m_stepYmm) * (m_indexedPositionsY.size() - 1) +
		FRAME_REL_H * firstImage.rows - 2 * (m_indexedPositionsX.size() - 1) * BIAS_Y_PIXELS);
	for (std::pair<float, size_t> indexedPositionZ : m_indexedPositionsZ)
	{
		float z = indexedPositionZ.first;
		Layer layer(z, m_rows, m_cols);
		m_layers.push_back(layer);
	}
}

void Map::stitchImages(const std::vector<std::vector<std::string>>& scanPositions, const std::vector<cv::Mat>& images)
{
	// Convert steps from mm to pixels and add preliminarly known biases if need
	const size_t stepPixelsX = mm2pixels(m_stepXmm) + BIAS_X_PIXELS;
	const size_t stepPixelsY = mm2pixels(m_stepYmm);

	// Positions by coordinates
	const std::vector<std::string>& positionsX = scanPositions[0];
	const std::vector<std::string>& positionsY = scanPositions[1];
	const std::vector<std::string>& positionsZ = scanPositions[2];

	// Store start position with initial margins for further calculation of capillaries positions
	m_startXmm = (float)atof(positionsX[0].c_str()) + pixels2mm((size_t)(MARGIN_REL_X * images[0].cols));
	m_startYmm = (float)atof(positionsY[0].c_str()) + pixels2mm((size_t)(MARGIN_REL_Y * images[0].rows));

	// For all positions and corresponding images
	for (size_t imageIndex = 0; imageIndex < images.size(); imageIndex++)
	{
		// Position of iterated source image
		float x = (float)atof(positionsX[imageIndex].c_str());
		float y = (float)atof(positionsY[imageIndex].c_str());
		float z = (float)atof(positionsZ[imageIndex].c_str());

		// Index is in flipped direction by X and in the same direction by Y
		size_t indexX = m_indexedPositionsX.size() - 1 - m_indexedPositionsX[x];
		size_t indexY = m_indexedPositionsY[y];

		// Calculate offsets in the desination image and store to skip unwanted corners on seams
		size_t dstOffsetX = stepPixelsX * indexX;
		size_t dstOffsetY = stepPixelsY * indexY + BIAS_Y_PIXELS * indexX;

		// Store all cols in kernel neighborhood to avoid false-positive corners around seams
		if ((dstOffsetX > 0) && !isOnSeam(dstOffsetX, false))
		{
			for (size_t col = dstOffsetX - DEEP_SMOOTHING_KERNEL_SIZE; col <= dstOffsetX + DEEP_SMOOTHING_KERNEL_SIZE; col++)
			{
				m_seamCols.push_back(col);
			}
		}

		// Store all rows in kernel neighborhood to avoid false-positive corners around seams
		if ((dstOffsetY > 0) && !isOnSeam(dstOffsetY, true))
		{
			for (size_t row = dstOffsetY - DEEP_SMOOTHING_KERNEL_SIZE; row <= dstOffsetY + DEEP_SMOOTHING_KERNEL_SIZE; row++)
			{
				m_seamRows.push_back(row);
			}
		}

		// Frame width is non-onerlapped vertical area for all frames before last or whole last frame
		size_t frameW = (indexX < m_indexedPositionsX.size() - 1) ?
			stepPixelsX :
			(size_t)(FRAME_REL_W * images[imageIndex].cols);

		// Frame height is non-onerlapped horizontal area for all frames before last or whole last frame
		size_t frameH = (indexY < m_indexedPositionsY.size() - 1) ?
			stepPixelsY :
			(size_t)(FRAME_REL_H * images[imageIndex].rows);

		// Select destination layer according to z
		size_t layerIndex = m_indexedPositionsZ[z];
		ByteMatrix& dstMatrix = m_layers[layerIndex].matrix;

		// Copy pixels from source to destination matrix
		stitchSingleImage(dstMatrix, images[imageIndex], dstOffsetX, dstOffsetY, frameW, frameH);
	}
}

void Map::stitchSingleImage(ByteMatrix& dstMatrix, const cv::Mat& srcImage, const size_t dstOffsetX, const size_t dstOffsetY,
	const size_t frameW, const size_t frameH)
{
	// Convert offsets from mm to pixel
	size_t srcOffsetRow = (size_t)(MARGIN_REL_Y * srcImage.rows);
	size_t srcOffsetCol = (size_t)(MARGIN_REL_X * srcImage.cols);

	// For all rows in the source frame
	for (size_t srcRow = 0; srcRow < frameH; srcRow++)
	{
		// Crop destination image
		int dstOffsetRow = (int)(dstOffsetY + srcRow) - (int)((m_indexedPositionsX.size() - 1) * BIAS_Y_PIXELS);
		if ((dstOffsetRow < 0) || (dstOffsetRow > m_rows - 1))
		{
			continue;
		}

		// Copy to destination image
		for (size_t srcCol = 0; srcCol < frameW; srcCol++)
		{
			byte val = srcImage.at<byte>((int)(srcOffsetRow + srcRow), (int)(srcOffsetCol + srcCol));
			dstMatrix.set((size_t)dstOffsetRow, dstOffsetX + srcCol, val);
		}
	}
}

void Map::copyScanPosFile(const std::string& scanPosFolderName, const std::string& outputFolderName)
{
	std::string scanPosSrcPathFilename = scanPosFolderName + "/" + SCAN_POS_FILENAME;
	std::string scanPosDstPathFilename = outputFolderName + "/Stitched/" + SCAN_POS_FILENAME;
	copyFile(scanPosSrcPathFilename, scanPosDstPathFilename);
}

// For debugging purpose only
void Map::markCorners(ByteMatrix& matrix, std::vector<ScoredCorner>& scoredCorners)
{
	// Number of pixels around the central pixel for valid marker odd sizes
	size_t halfMarkerSize = m_markerSize / 2;

	for (const ScoredCorner& scoredCorner : scoredCorners)
	{
		// Convert position of the corner from mm to pixels
		size_t cornerRow = mm2pixels(scoredCorner.y);
		size_t cornerCol = mm2pixels(scoredCorner.x);

		// Limit marker boundaries to not excess the matrix
		size_t markerRowMin = std::max<size_t>(cornerRow - halfMarkerSize, 0);
		size_t markerRowMax = std::min<size_t>(cornerRow + halfMarkerSize, matrix.rows() - 1);
		size_t markerColMin = std::max<size_t>(cornerCol - halfMarkerSize, 0);
		size_t markerColMax = std::min<size_t>(cornerCol + halfMarkerSize, matrix.cols() - 1);

		// Paint over the marker
		for (size_t row = markerRowMin; row <= markerRowMax; row++)
		{
			for (size_t col = markerColMin; col <= markerColMax; col++)
			{
				matrix.set(row, col, WHITE);
			}
		}
	}
}
