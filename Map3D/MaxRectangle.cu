#include "hip/hip_runtime.h"
#include "UtilsCUDA.h"
#include "MaxRectangle.h"

/*
	Kernel Device (GPU) variables and functions
	===========================================
*/

__device__ constexpr float PI() { return 3.14159265F; }

__global__ void resetRotatedCapillary(byte* d_dstMatrix, int dstRows, int dstCols)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x >= dstCols) || (y >= dstRows))
	{
		return;
	}

	d_dstMatrix[y * dstCols + x] = LIGHT_GRAY;
}

__global__ void performCapillaryRotation(byte* d_srcMatrix, byte* d_dstMatrix,
	int srcRows, int srcCols, int dstRows, int dstCols,
	float srcCenterX, float srcCenterY, float dstCenterX, float dstCenterY, float angle)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x >= srcCols) || (y >= srcRows))
	{
		return;
	}

	byte pixel = d_srcMatrix[y * srcCols + x];
	if (pixel != WHITE)
	{
		return;
	}

	float deltaX = srcCenterX - (float)x;
	float deltaY = srcCenterY - (float)y;
	float radius = sqrtf(deltaX * deltaX + deltaY * deltaY);
	float angleSrc = atan2f(deltaY, deltaX);
	float angleDst = angleSrc + angle + PI(); // axis Y is counter-directional to rows numeration
	size_t dstX = (size_t)((int)dstCenterX + (int)round(radius * cosf(angleDst)));
	size_t dstY = (size_t)((int)dstCenterY + (int)round(radius * sinf(angleDst)));
	d_dstMatrix[dstY * dstCols + dstX] = pixel;
}

/*
	Public Host (CPU) functions to call kernel Device (GPU) functions
	=================================================================
*/

MaxRectangle::MaxRectangle(ByteMatrix& byteMatrix, PixelPos start, size_t rows, size_t cols,
	const std::string& layerFolderName, size_t capillaryIndex)
{
	// Create and fill original rectangle
	m_originalCapillary = ByteMatrix(rows, cols);
	for (size_t row = 0; row < rows; row++)
	{
		for (size_t col = 0; col < cols; col++)
		{
			byte pixel = byteMatrix.get(start.pixelRow + row, start.pixelCol + col);
			m_originalCapillary.set(row, col, pixel);
		}
	}

#ifdef _DEBUG
	// Save image of original capillary
	std::string capillaryFilename = layerFolderName + "/Capillary" +
		std::to_string(capillaryIndex + 1) + ".bmp";
	cv::imwrite(capillaryFilename, m_originalCapillary.asCvMatU8());
#endif
	// Prepare byte matrices for rotated and dilated rectangle - large enough for any rotation
	size_t rotatedSize = 2 * std::max(rows, cols);
	m_rotatedCapillary = ByteMatrix(rotatedSize, rotatedSize);
	m_dilatedCapillary = ByteMatrix(rotatedSize, rotatedSize);

	// Calculate center of updated rectangle which is the same as center of original rectangle
	size_t centralRow = start.pixelRow + rows / 2;
	size_t centralCol = start.pixelCol + cols / 2;
	m_centerInImage = PixelPos(centralRow, centralCol);

	m_limitUp = 0;
	m_limitDn = 0;
	m_limitLf = 0;
	m_limitRt = 0;

	m_rowFrameInRotated = 0;
	m_colFrameInRotated = 0;
	m_foundAngleRadians = 0.0F;
	m_score = 0.0F;
}

std::vector<PixelPos> MaxRectangle::findRectangle(const std::string& layerFolderName, size_t capillaryIndex)
{
	std::vector<PixelPos> rotatedRectangle;
	size_t angleDegrees = 0;
	bool foundInscribedRectangle = false;
	for (; angleDegrees < 180; angleDegrees += 10)
	{
		rotateCapillary(angleDegrees);
		findCapillaryLimits();
		dilateRotatedCapillary();
		foundInscribedRectangle = findInscribedRectangle();
		if (foundInscribedRectangle)
		{
			break;
		}
	}

#ifdef _DEBUG
	// Save image of found rotated capillary
	std::string rotatedFilename = layerFolderName + "/Rotated" +
		std::to_string(capillaryIndex + 1) + ".bmp";
	cv::imwrite(rotatedFilename, m_rotatedCapillary.asCvMatU8());

	// Frame is marked only if inscribed rectangle is found
	markFrameInDilatedCapillary(foundInscribedRectangle);

	// Save image of found dilated capillary with frame
	std::string dilatedFilename = layerFolderName + "/Dilated" +
		std::to_string(capillaryIndex + 1) + ".bmp";
	cv::imwrite(dilatedFilename, m_dilatedCapillary.asCvMatU8());
#endif
	if (!foundInscribedRectangle)
	{
		return rotatedRectangle;
	}

	// Width map of the capillary along the longest axis of symmetry of the rectangle
	writeWidthMap(layerFolderName, capillaryIndex);

	// Convert found angle to radians and flip sign: rotated frame on fixed capillary
	m_foundAngleRadians = -deg2rad(angleDegrees);

	rotatedRectangle = getRotatedRectangle();
	return rotatedRectangle;
}

float MaxRectangle::getAngle()
{
	return m_foundAngleRadians;
}

float MaxRectangle::getScore()
{
	return 100.0F * (m_score - SCORE_THRESHOLD);
}

/*
	Private Host (CPU) functions to call kernel Device (GPU) functions
	==================================================================
*/

void MaxRectangle::rotateCapillary(size_t angleDegrees)
{
	// Convert given angle to radians
	float angle = deg2rad(angleDegrees);

	// Sizes of source and destination
	size_t rowsSrc = m_originalCapillary.rows();
	size_t colsSrc = m_originalCapillary.cols();
	size_t rowsDst = m_rotatedCapillary.rows();
	size_t colsDst = m_rotatedCapillary.cols();

	// Centers of source and destination
	size_t centerRowSrc = rowsSrc / 2;
	size_t centerColSrc = colsSrc / 2;
	size_t centerRowDst = rowsDst / 2;
	size_t centerColDst = colsDst / 2;

	// Allocate device memory buffers
	byte* d_srcBuffer = nullptr;
	checkCuda(hipMalloc(&d_srcBuffer, rowsSrc * colsSrc));
	byte* d_dstBuffer = nullptr;
	checkCuda(hipMalloc(&d_dstBuffer, rowsDst * colsDst));

	// Parameters to launch parallel threads
	dim3 blockSize(128, 1);
	dim3 numBlocksSrc(divideCeil((int)colsSrc, blockSize.x), divideCeil((int)rowsSrc, blockSize.y), 1);
	dim3 numBlocksDst(divideCeil((int)colsDst, blockSize.x), divideCeil((int)rowsDst, blockSize.y), 1);

	// Fill source device buffer by original capillary
	checkCuda(hipMemcpy(d_srcBuffer, m_originalCapillary.getBuffer(), rowsSrc * colsSrc, hipMemcpyHostToDevice));

	// Fill background of rotated capillary
	resetRotatedCapillary<<<numBlocksDst, blockSize>>>(d_dstBuffer, (int)rowsDst, (int)colsDst);

	// Calculate rotated capillary on GPU
	performCapillaryRotation<<<numBlocksSrc, blockSize>>>(d_srcBuffer, d_dstBuffer,
		(int)rowsSrc, (int)colsSrc, (int)rowsDst, (int)colsDst,
		(float)centerColSrc, (float)centerRowSrc, (float)centerColDst, (float)centerRowDst, angle);
	checkCuda(hipDeviceSynchronize());

	// Get calculated rotated capillary from device memory and free it
	checkCuda(hipMemcpy(m_rotatedCapillary.getBuffer(), d_dstBuffer, rowsDst * colsDst, hipMemcpyDeviceToHost));
	checkCuda(hipFree(d_dstBuffer));
}

void MaxRectangle::findCapillaryLimits()
{
	// Used to break nested loops
	bool found;

	// Find limit: Up
	found = false;
	for (size_t row = 0; (row < m_rotatedCapillary.rows()) && !found; row++)
	{
		for (size_t col = 0; (col < m_rotatedCapillary.cols()) && !found; col++)
		{
			byte pixel = m_rotatedCapillary.get(row, col);
			if (pixel == WHITE)
			{
				m_limitUp = row;
				found = true;
			}
		}
	}

	// Find limit: Rt
	found = false;
	for (size_t col = m_rotatedCapillary.cols() - 1; (col > 0) && !found; col--)
	{
		for (size_t row = 0; (row < m_rotatedCapillary.rows()) && !found; row++)
		{
			byte pixel = m_rotatedCapillary.get(row, col);
			if (pixel == WHITE)
			{
				m_limitRt = col;
				found = true;
			}
		}
	}

	// Find limit: Dn
	found = false;
	for (size_t row = m_rotatedCapillary.rows() - 1; (row > 0) && !found; row--)
	{
		for (size_t col = m_rotatedCapillary.cols() - 1; (col > 0) && !found; col--)
		{
			byte pixel = m_rotatedCapillary.get(row, col);
			if (pixel == WHITE)
			{
				m_limitDn = row;
				found = true;
			}
		}
	}

	// Find limit: Lf
	found = false;
	for (size_t col = 0; (col < m_rotatedCapillary.cols()) && !found; col++)
	{
		for (size_t row = m_rotatedCapillary.rows() - 1; (row > 0) && !found; row--)
		{
			byte pixel = m_rotatedCapillary.get(row, col);
			if (pixel == WHITE)
			{
				m_limitLf = col;
				found = true;
			}
		}
	}
}

void MaxRectangle::dilateRotatedCapillary()
{
	const size_t dilationKernelSize = 3;
	size_t halfKernelSize = dilationKernelSize / 2;
	size_t threshold = dilationKernelSize * dilationKernelSize / 2 - 1;

	// Reset previous dilation
	m_dilatedCapillary.clean();

	for (size_t row = m_limitUp; row <= m_limitDn; row++)
	{
		for (size_t col = m_limitLf; col <= m_limitRt; col++)
		{
			// Count pixels in kernel
			size_t numWhitePixelsInKernel = 0;
			for (size_t kernelRow = row - halfKernelSize; kernelRow <= row + halfKernelSize; kernelRow++)
			{
				for (size_t kernelCol = col - halfKernelSize; kernelCol <= col + halfKernelSize; kernelCol++)
				{
					if (m_rotatedCapillary.get(kernelRow, kernelCol) == WHITE)
					{
						numWhitePixelsInKernel++;
					}
				}
			}

			// Skip empty areas
			if (numWhitePixelsInKernel < threshold)
			{
				continue;
			}

			// Fill underfilled areas
			for (size_t kernelRow = row - halfKernelSize; kernelRow <= row + halfKernelSize; kernelRow++)
			{
				for (size_t kernelCol = col - halfKernelSize; kernelCol <= col + halfKernelSize; kernelCol++)
				{
					m_dilatedCapillary.set(kernelRow, kernelCol, WHITE);
				}
			}
		}
	}
}

bool MaxRectangle::findInscribedRectangle()
{
	for (size_t row = m_limitUp; row <= m_limitDn - FRAME_HEIGHT; row++)
	{
		for (size_t col = m_limitLf; col <= m_limitRt - FRAME_WIDTH; col++)
		{
			size_t numWhitePixelsInRectangle = 0;
			for (size_t frameRow = row; frameRow < row + FRAME_HEIGHT; frameRow++)
			{
				for (size_t frameCol = col; frameCol < col + FRAME_WIDTH; frameCol++)
				{
					if (m_dilatedCapillary.get(frameRow, frameCol) == WHITE)
					{
						numWhitePixelsInRectangle++;
					}
				}
			}

			float score = (float)numWhitePixelsInRectangle / FRAME_WIDTH / FRAME_HEIGHT;
			if (score > m_score)
			{
				m_rowFrameInRotated = row;
				m_colFrameInRotated = col;
				m_score = score;
			}
		}
	}

	return m_score >= SCORE_THRESHOLD;
}

void MaxRectangle::markFrameInDilatedCapillary(bool foundInscribedRectangle)
{
	if (!foundInscribedRectangle)
	{
		return;
	}

	for (size_t row = m_rowFrameInRotated; row <= m_rowFrameInRotated + FRAME_HEIGHT; row++)
	{
		m_dilatedCapillary.set(row, m_colFrameInRotated, BLACK);
		m_dilatedCapillary.set(row, m_colFrameInRotated + FRAME_WIDTH, BLACK);
	}

	for (size_t col = m_colFrameInRotated; col <= m_colFrameInRotated + FRAME_WIDTH; col++)
	{
		m_dilatedCapillary.set(m_rowFrameInRotated, col, BLACK);
		m_dilatedCapillary.set(m_rowFrameInRotated + FRAME_HEIGHT, col, BLACK);
	}
}

void MaxRectangle::writeWidthMap(const std::string& layerFolderName, size_t capillaryIndex)
{
	std::string filenameWidthMap = layerFolderName + "/WidthCapillary" +
		std::to_string(capillaryIndex + 1) + ".csv";
	std::ofstream fileWidthMap(filenameWidthMap);
	fileWidthMap << "Distance mm,Width mm" << std::endl;

	for (size_t row = m_limitUp; row <= m_limitDn; row++)
	{
		float distance = pixels2mm(row - m_limitUp);
		size_t widthPixels = 0;
		for (size_t col = m_limitLf; col <= m_limitRt; col++)
		{
			widthPixels += m_dilatedCapillary.get(row, col) == WHITE ? 1 : 0;
		}
		float width = pixels2mm(widthPixels);
		fileWidthMap <<
			std::setw(8) << distance << "," <<
			std::setw(8) << width << std::endl;
	}

	fileWidthMap.close();
}

std::vector<PixelPos> MaxRectangle::getRotatedRectangle()
{
	size_t centerRow = m_dilatedCapillary.rows() / 2;
	size_t centerCol = m_dilatedCapillary.cols() / 2;

	// Set vertices of original unrotated rectangle in predefined order
	std::vector<PixelPos> originalRectangle;
	originalRectangle.push_back(PixelPos(m_rowFrameInRotated, m_colFrameInRotated));
	originalRectangle.push_back(PixelPos(m_rowFrameInRotated, m_colFrameInRotated + FRAME_WIDTH));
	originalRectangle.push_back(PixelPos(m_rowFrameInRotated + FRAME_HEIGHT, m_colFrameInRotated + FRAME_WIDTH));
	originalRectangle.push_back(PixelPos(m_rowFrameInRotated + FRAME_HEIGHT, m_colFrameInRotated));

	std::vector<PixelPos> rotatedRectangle;
	for (const PixelPos& pixelPos : originalRectangle)
	{
		float deltaY = (float)centerRow - (float)pixelPos.pixelRow;
		float deltaX = (float)centerCol - (float)pixelPos.pixelCol;
		float radius = std::sqrtf(deltaX * deltaX + deltaY * deltaY);
		float angleSrc = std::atan2f(deltaY, deltaX);
		float angleDst = makeCentrosymmetric(angleSrc + m_foundAngleRadians); // axis Y is counter-directional to rows numeration
		size_t rowInImage = (size_t)((int)m_centerInImage.pixelRow + (int)std::round(radius * std::sinf(angleDst)));
		size_t colInImage = (size_t)((int)m_centerInImage.pixelCol + (int)std::round(radius * std::cosf(angleDst)));
		rotatedRectangle.push_back(PixelPos(rowInImage, colInImage));
	}

	return rotatedRectangle;
}
