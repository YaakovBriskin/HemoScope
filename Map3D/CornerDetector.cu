#include "hip/hip_runtime.h"
#include "Utils.h"
#include "UtilsCUDA.h"
#include "CornerDetector.h"

/*
	Kernel Device (GPU) variables and functions
	===========================================
*/

__global__ void applySobelKernel(byte* d_srcMatrix, byte* d_dstMatrix,
	int rows, int cols, bool isByX)
{
	const short kernelGx[CORNER_DETECTION_KERNEL_SIZE * CORNER_DETECTION_KERNEL_SIZE] =
	{
		-1, 0, 1,
		-2, 0, 2,
		-1, 0, 1
	};

	const short kernelGy[CORNER_DETECTION_KERNEL_SIZE * CORNER_DETECTION_KERNEL_SIZE] =
	{
		-1, -2, -1,
		 0,  0,  0,
		 1,  2,  1
	};

	// Number of pixels around the central pixel for valid kernel odd sizes: 3, 5, 7
	const int halfKernelSize = (int)CORNER_DETECTION_KERNEL_SIZE / 2;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x >= cols) || (y >= rows))
	{
		return;
	}

	// Skip margins with zeroing of result
	if ((x < halfKernelSize) ||
		(x >= cols - halfKernelSize) ||
		(y < halfKernelSize) ||
		(y >= rows - halfKernelSize))
	{
		d_dstMatrix[y * cols + x] = 0;
		return;
	}

	short* kernel = isByX ? (short*)kernelGx : (short*)kernelGy;

	// Calculate convolution with the kernel
	short convolved = 0;
	for (int kernelY = 0; kernelY < CORNER_DETECTION_KERNEL_SIZE; kernelY++)
	{
		for (int kernelX = 0; kernelX < CORNER_DETECTION_KERNEL_SIZE; kernelX++)
		{
			int matrixY = y - halfKernelSize + kernelY;
			int matrixX = x - halfKernelSize + kernelX;
			short matrixVal = (short)d_srcMatrix[matrixY * cols + matrixX];
			short kernelVal = kernel[kernelY * CORNER_DETECTION_KERNEL_SIZE + kernelX];
			convolved += matrixVal * kernelVal;
		}
	}

	// Trim convolution result before setting to destination matrix
	if (convolved < 0)
	{
		convolved = -convolved;
	}
	if (convolved > (short)WHITE)
	{
		convolved = (short)WHITE;
	}
	d_dstMatrix[y * cols + x] = (byte)convolved;
}

__global__ void combineSobelFilters(byte* d_srcMatrixGx, byte* d_srcMatrixGy,
	byte* d_dstMatrix, int rows, int cols)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x >= cols) || (y >= rows))
	{
		return;
	}

	short valGx = (short)d_srcMatrixGx[y * cols + x];
	short valGy = (short)d_srcMatrixGy[y * cols + x];
	short val = valGx + valGy;
	if (val > (short)WHITE)
	{
		val = (short)WHITE;
	}
	d_dstMatrix[y * cols + x] = (byte)val;
}

/*
	Public Host (CPU) functions to call kernel Device (GPU) functions
	=================================================================
*/

CornerDetector::CornerDetector()
{
	m_z = 0.0F;
	m_croppedRows = 400;
	m_gradientThreshold = 35;
	m_minDistancePixels = 200;

	md_srcBuffer = nullptr;
	md_dstBufferSobelGx = nullptr;
	md_dstBufferSobelGy = nullptr;
	md_dstBufferSobel = nullptr;
}

CornerDetector::~CornerDetector()
{
	//freeDeviceBuffers();
}

void CornerDetector::setLayerPosition(float z)
{
	m_z = z;
}

/*
	Apply Gx and Gy Sobel kernels and find average gradient values over predefined threshold
*/
std::vector<ScoredCorner> CornerDetector::getCornersSobel(Map& map, ByteMatrix& matrix,
	const std::string& capillariesFolderName, size_t layerIndex)
{
	// Number of pixels around the central pixel for valid kernel odd sizes: 3, 5, 7
	size_t halfKernelSize = CORNER_DETECTION_KERNEL_SIZE / 2;

	// Filled and returned detected corners
	std::vector<ScoredCorner> scoredCorners;

	int rows = (int)matrix.rows();
	int cols = (int)matrix.cols();

	// Allocate device memory buffers and fill source device buffer by processed matrix
	allocateDeviceBuffers(rows, cols);
	checkCuda(hipMemcpy(md_srcBuffer, matrix.getBuffer(), rows * cols, hipMemcpyHostToDevice));

	// Parameters to launch parallel threads
	dim3 blockSize(128, 1);
	dim3 numBlocks(divideCeil(cols, blockSize.x), divideCeil(rows, blockSize.y), 1);

	applySobelKernel<<<numBlocks, blockSize>>>(md_srcBuffer, md_dstBufferSobelGx,
		rows, cols, true);
	applySobelKernel<<<numBlocks, blockSize>>>(md_srcBuffer, md_dstBufferSobelGy,
		rows, cols, false);
	combineSobelFilters<<<numBlocks, blockSize>>>(md_dstBufferSobelGx, md_dstBufferSobelGy,
		md_dstBufferSobel, rows, cols);
	checkCuda(hipDeviceSynchronize());

	ByteMatrix gradient = ByteMatrix(rows, cols);
	checkCuda(hipMemcpy(gradient.getBuffer(), md_dstBufferSobel, rows * cols, hipMemcpyDeviceToHost));
	//cv::Mat y3 = cv::Mat(rows, cols, CV_8U, gradient.getBuffer());
	//cv::imwrite("D:/tmp/Gradient.bmp", gradient.asCvMatU8());

	for (size_t row = halfKernelSize; row < rows - m_croppedRows - halfKernelSize; row++)
	{
		for (size_t col = halfKernelSize; col < cols - halfKernelSize; col++)
		{
			// Skip possible false-positive corners on seams
			if (map.isOnSeam(row, true) || map.isOnSeam(col, false))
			{
				continue;
			}

			// Skip unexpected gray levels - mainly on flares
			byte grayLevel = matrix.get(row, col);
			if (!isValidGrayLevelOriginal(grayLevel))
			{
				continue;
			}

			// Calculate the average value of gradients in the kernel
			unsigned short sumGrad = 0;
			for (short kernelRow = -1; kernelRow <= 1; kernelRow++)
			{
				for (short kernelCol = -1; kernelCol <= 1; kernelCol++)
				{
					sumGrad += (unsigned short)gradient.get(row + kernelRow, col + kernelCol);
				}
			}
			byte avgGrad = (byte)std::roundf((float)sumGrad /
				CORNER_DETECTION_KERNEL_SIZE / CORNER_DETECTION_KERNEL_SIZE);

			// If the gradient does not meet quality criteria - skip examined pixel
			if (avgGrad < m_gradientThreshold)
			{
				continue;
			}

			// Iterate previously accumulated corners to whehter near corner was added before
			bool nearCornerFound = false;
			size_t cornerIndex = 0;
			for (; cornerIndex < scoredCorners.size(); cornerIndex++)
			{
				int distPixelsX = (int)mm2pixels(scoredCorners[cornerIndex].x) - (int)col;
				int distPixelsY = (int)mm2pixels(scoredCorners[cornerIndex].y) - (int)row;
				size_t distPixels2 = (size_t)(distPixelsX * distPixelsX + distPixelsY * distPixelsY);
				if (distPixels2 < m_minDistancePixels * m_minDistancePixels)
				{
					nearCornerFound = true;
					break;
				}
			}

			// Parameters of overwritten near corner (if found) or accumulated corner
			float x = pixels2mm(col);
			float y = pixels2mm(row);
			float nomalizedScore = 100.0F * ((float)avgGrad / m_gradientThreshold - 1.0F);

			if (nearCornerFound)
			{
				// If near corner is found - overwrite by current corner if it has better score
				if (nomalizedScore > scoredCorners[cornerIndex].score)
				{
					scoredCorners[cornerIndex].x = x;
					scoredCorners[cornerIndex].y = y;
					scoredCorners[cornerIndex].score = nomalizedScore;
					scoredCorners[cornerIndex].grayLevel = grayLevel;
				}
			}
			else
			{
				// Accumulate new corner
				ScoredCorner scoredCorner(x, y, m_z, nomalizedScore, grayLevel);
				scoredCorners.push_back(scoredCorner);
			}
		}
	}

	// Sort found corners by score in descending order
	std::sort(scoredCorners.begin(), scoredCorners.end(), [](ScoredCorner cornerL, ScoredCorner cornerR) {
		return cornerL.score > cornerR.score;
	});

#ifdef _DEBUG
	std::string filenameGradient = capillariesFolderName + "/Gradient" + std::to_string(layerIndex + 1) + ".bmp";
	cv::imwrite(filenameGradient, gradient.asCvMatU8());
	std::string filenameLayer = capillariesFolderName + "/Layer" + std::to_string(layerIndex + 1) + ".csv";
	writeCorners(scoredCorners, filenameLayer);
#endif
	return scoredCorners;
}

/*
	Private Host (CPU) functions to call kernel Device (GPU) functions
	==================================================================
*/

void CornerDetector::allocateDeviceBuffers(size_t rows, size_t cols)
{
	if (md_srcBuffer == nullptr)
	{
		checkCuda(hipMalloc(&md_srcBuffer, rows * cols));
	}

	if (md_dstBufferSobelGx == nullptr)
	{
		checkCuda(hipMalloc(&md_dstBufferSobelGx, rows * cols));
	}

	if (md_dstBufferSobelGy == nullptr)
	{
		checkCuda(hipMalloc(&md_dstBufferSobelGy, rows * cols));
	}

	if (md_dstBufferSobel == nullptr)
	{
		checkCuda(hipMalloc(&md_dstBufferSobel, rows * cols));
	}
}

void CornerDetector::freeDeviceBuffers()
{
	if (md_srcBuffer != nullptr)
	{
		checkCuda(hipFree(md_srcBuffer));
	}

	if (md_dstBufferSobelGx != nullptr)
	{
		checkCuda(hipFree(md_dstBufferSobelGx));
	}

	if (md_dstBufferSobelGy != nullptr)
	{
		checkCuda(hipFree(md_dstBufferSobelGy));
	}

	if (md_dstBufferSobel != nullptr)
	{
		checkCuda(hipFree(md_dstBufferSobel));
	}
}

void CornerDetector::writeCorners(const std::vector<ScoredCorner>& scoredCorners, const std::string& filenameLayer)
{
	std::ofstream fileLayer(filenameLayer);
	fileLayer << "Num,Row,Col,Gray level,Score" << std::endl;
	size_t number = 1;
	for (const ScoredCorner& scoredCorner : scoredCorners)
	{
		size_t row = mm2pixels(scoredCorner.y);
		size_t col = mm2pixels(scoredCorner.x);
		fileLayer <<
			number++ << "," <<
			row << "," <<
			col << "," <<
			(int)scoredCorner.grayLevel << "," <<
			std::setprecision(4) << scoredCorner.score << std::endl;
	}
	fileLayer.close();
}
